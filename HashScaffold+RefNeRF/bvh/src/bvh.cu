#include "hip/hip_runtime.h"
#ifndef BVH_BVH_CU
#define BVH_BVH_CU
#include <tuple>
#include "bvh.h"
#include "construct.cuh"
#include "trace.cuh"

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
create_bvh(const torch::Tensor& means3D, const torch::Tensor& scales, const torch::Tensor& rotations, const torch::Tensor& nodes, const torch::Tensor& aabbs){
    const uint32_t P = means3D.size(0);

    auto int_opts = means3D.options().dtype(torch::kInt32);
    auto float_opts = means3D.options().dtype(torch::kFloat32);

    torch::Tensor mortons = torch::zeros({P}, means3D.options().dtype(torch::kLong));

    construct_bvh(
            P,
            means3D.contiguous().data_ptr<float>(),
            scales.contiguous().data_ptr<float>(),
            rotations.contiguous().data_ptr<float>(),
            (int32_t*)nodes.contiguous().data_ptr<int>(),
            aabbs.contiguous().data_ptr<float>(),
            (uint64_t*)mortons.contiguous().data_ptr<int64_t>()
    );
    return std::make_tuple(nodes, aabbs, mortons);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
trace_bvh(const torch::Tensor& nodes, const torch::Tensor& aabbs,
          const torch::Tensor& rays_o, const torch::Tensor& rays_d,
          const torch::Tensor& means3D, const torch::Tensor& covs3D,
          const torch::Tensor& opacities){
    int32_t num_rays = rays_o.size(0);

    auto int_opts = rays_o.options().dtype(torch::kInt32);
    auto float_opts = rays_o.options();
    torch::Tensor num_contributes = torch::zeros({num_rays, 1}, int_opts);

    auto result = trace_bvh_cuda(num_rays,
                   nodes.contiguous().data_ptr<int32_t>(),
                   aabbs.contiguous().data_ptr<float>(),
                   (float3*)rays_o.contiguous().data_ptr<float>(),
                   (float3*)rays_d.contiguous().data_ptr<float>(),
                   (float3*)means3D.contiguous().data_ptr<float>(),
                   covs3D.contiguous().data_ptr<float>(),
                   opacities.contiguous().data_ptr<float>(),
                   num_contributes.contiguous().data_ptr<int32_t>());

//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);
//     hipEventRecord(start);
//     float milliseconds = 0;

    int32_t num_rendered = std::get<0>(result);
    thrust::device_vector<int32_t>& point_list_vec = std::get<1>(result);
    thrust::device_vector<float3>& position_list_vec = std::get<2>(result);
    thrust::device_vector<int32_t>& ray_id_list_vec = std::get<3>(result);
    if (num_rendered == 0){
        torch::Tensor point_list_tensor = torch::zeros({0, 1}, int_opts);
        torch::Tensor position_list_tensor = torch::zeros({0, 3}, float_opts);
        torch::Tensor ray_id_list_tensor = torch::zeros({0, 3}, float_opts);
        return std::make_tuple(num_contributes, point_list_tensor, position_list_tensor, ray_id_list_tensor);
    }
    int32_t* point_list = thrust::raw_pointer_cast(point_list_vec.data());
    int32_t size = point_list_vec.size();
    torch::Tensor point_list_tensor = torch::from_blob(point_list, {size, 1}, int_opts);
    point_list_tensor = point_list_tensor.clone();

    float* position_list = (float*)thrust::raw_pointer_cast(position_list_vec.data());
    torch::Tensor position_list_tensor = torch::from_blob(position_list, {size, 3}, float_opts);
    position_list_tensor = position_list_tensor.clone();

    int32_t* ray_id_list = thrust::raw_pointer_cast(ray_id_list_vec.data());
    torch::Tensor ray_id_list_tensor = torch::from_blob(ray_id_list, {size, 1}, int_opts);
    ray_id_list_tensor = ray_id_list_tensor.clone();

//     hipEventRecord(stop);
//     hipEventSynchronize(stop);
//     hipEventElapsedTime(&milliseconds, start, stop);
//     std::cout << "after time: " << milliseconds << " ms" << std::endl;
//     hipEventRecord(start);
    return std::make_tuple(num_contributes, point_list_tensor, position_list_tensor, ray_id_list_tensor);
}


std::tuple<torch::Tensor, torch::Tensor>
trace_bvh_opacity(const torch::Tensor& nodes, const torch::Tensor& aabbs,
          const torch::Tensor& rays_o, const torch::Tensor& rays_d,
          const torch::Tensor& means3D, const torch::Tensor& covs3D,
          const torch::Tensor& opacities, const torch::Tensor& normals){
    int32_t num_rays = rays_o.numel() / rays_o.size(-1);
    auto rays_o_shape = rays_o.sizes().slice(0, rays_o.dim() - 1);
//     auto rays_o_shape = rays_o.sizes().vec();
//     rays_o_shape.pop_back();
//     rays_o_shape.push_back(1);

    auto int_opts = rays_o.options().dtype(torch::kInt32);
    auto float_opts = rays_o.options();
    torch::Tensor num_contributes = torch::zeros(rays_o_shape, int_opts);
    torch::Tensor rendered_opacity = torch::ones(rays_o_shape, float_opts);

    trace_bvh_opacity_cuda(num_rays,
                   nodes.contiguous().data_ptr<int32_t>(),
                   aabbs.contiguous().data_ptr<float>(),
                   (float3*)rays_o.contiguous().data_ptr<float>(),
                   (float3*)rays_d.contiguous().data_ptr<float>(),
                   (float3*)means3D.contiguous().data_ptr<float>(),
                   covs3D.contiguous().data_ptr<float>(),
                   opacities.contiguous().data_ptr<float>(),
                   (float3*)normals.contiguous().data_ptr<float>(),
                   num_contributes.contiguous().data_ptr<int32_t>(),
                   rendered_opacity.contiguous().data_ptr<float>());
    return std::make_tuple(num_contributes, rendered_opacity);
}

#endif //BVH_BVH_CU